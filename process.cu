#include "hip/hip_runtime.h"
#include <NvInfer.h>
#include <iostream>
#include "cuda_utils.h"

namespace Process
{
    __global__ void add_device(int* a, int* b)
    {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        b[index] = a[index] + 3;
    }

    void add(int* a, int* b)
    {
        int* a_d;
        int* b_d;
        CUDA_CHECK(hipMalloc(&a_d, 4 * sizeof(int)));
        CUDA_CHECK(hipMalloc(&b_d, 4 * sizeof(int)));
        CUDA_CHECK(hipMemcpy(a_d, a, 4 * sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(b_d, b, 4 * sizeof(int), hipMemcpyHostToDevice));
        add_device <<< 2, 2 >>> (a_d, b_d);
        CUDA_CHECK(hipMemcpy(a, a_d, 4 * sizeof(int), hipMemcpyDeviceToHost));
        CUDA_CHECK(hipMemcpy(b, b_d, 4 * sizeof(int), hipMemcpyDeviceToHost));
    }
}

extern "C"
{
    void add_c(int* a, int* b)
    {
        Process::add(a, b);
    }
}